#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void gTest(float* a){

 a[threadIdx.x+blockDim.x*blockIdx.x]=
(float)(threadIdx.x+blockDim.x*blockIdx.x);

}

int main(){
	float *da, *ha;
	int num_of_blocks=10, threads_per_block=32;
	int N=num_of_blocks*threads_per_block;
	ha=(float*)calloc(N, sizeof(float));
	hipMalloc((void**)&da, N*sizeof(float));
	gTest<<<dim3(num_of_blocks),
	dim3(threads_per_block)>>>(da);
	hipDeviceSynchronize();
	hipMemcpy(ha,da,N*sizeof(float),
	hipMemcpyDeviceToHost);
	for(int i=0;i<N;i++)
	printf("%g\n", ha[i]);
	free(ha);
	hipFree(da);
	return 0;
}


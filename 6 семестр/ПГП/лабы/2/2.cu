#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <fstream> 

using namespace std;

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat=value;\
	if(_m_cudaStat!=hipSuccess){\
		fprintf(stderr,"Error %s at line %d in file %s\n",hipGetErrorString(_m_cudaStat),__LINE__,__FILE__);\
		exit(1);\
	}\
}

__global__ void Vector(float* a, float* b, float* c, int N) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < N){
		a[i] = i + 1;
		b[i] = i + 1;
		c[i] = a[i] + b[i];
	}
	else {
		return;
	}
}

int main() {
	int N;
	float *da, *db, *dc, *a, *b, *c;
	float sum = 0, MEGA_sum=0;
	ofstream benchmark_output;
	benchmark_output.open("bench result.csv", ios_base::app);
	benchmark_output << "ThreadsPerBlock;Block;Timer" << endl;

	for (int k = 1 << 1; k <= 1 << 10; k = k << 1) {
		sum = 0;
		fprintf(stderr, "Threads per Blocks(%i):\n", k);
		for (int j = 10; j <= 23; j++)
		{
			N = 1 << j;
			a = (float*)calloc(N, sizeof(float));
			b = (float*)calloc(N, sizeof(float));
			c = (float*)calloc(N, sizeof(float));

			for (int i = 0; i < N; i++) {}

			CUDA_CHECK_RETURN(hipMalloc((void**)&da, N * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void**)&db, N * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void**)&dc, N * sizeof(float)));

			float elapsedTime;

			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			Vector <<< N / k, k >>> (da, db, dc, N);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);
			fprintf(stderr, "%f\n", elapsedTime);
			sum += elapsedTime;
			CUDA_CHECK_RETURN(hipGetLastError());
			CUDA_CHECK_RETURN(hipMemcpy(a, da, N * sizeof(float), hipMemcpyDeviceToHost));
			CUDA_CHECK_RETURN(hipMemcpy(b, db, N * sizeof(float), hipMemcpyDeviceToHost));
			CUDA_CHECK_RETURN(hipMemcpy(c, dc, N * sizeof(float), hipMemcpyDeviceToHost));
			hipEventDestroy(start);
			hipEventDestroy(stop);
			free(a);
			free(b);
			free(c);
			hipFree(da);
			hipFree(db);
			hipFree(dc);
			benchmark_output << k << ";" << N << ";" << elapsedTime << endl;
		}
		fprintf(stderr, "\nSummary time: %f\n\n", sum);
		MEGA_sum += sum;
	}
	benchmark_output.close();
	fprintf(stderr, "\nMEGA Summary time: %f\n\n", MEGA_sum);
	return 0;
}


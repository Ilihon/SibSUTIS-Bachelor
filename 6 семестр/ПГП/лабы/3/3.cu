#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define CUDA_CHECK_RETURN(value){\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess){\
		fprintf(stderr, "Error %s atline %d in file %s\n",\
		 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}\
}\

__global__ void gTest1(float* a){
 	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int I = gridDim.x * blockDim.x;
	a[i + j * I] = (float)(i + j * I);
}

__global__ void gTest2(float *a){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int J = gridDim.y * blockDim.y;
	a[j + i * J] = (float)(threadIdx.y + blockIdx.y * blockDim.y);
}

__global__ void transpos(float *a, float *b){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int I = gridDim.x * blockDim.x;
	b[j + i * I] = a[i + j * I];
}
 
int main(){
 	float elapsedTime;
	hipEvent_t start, stop;
	float *da, *ha, *db, *hb;

 	int num_of_blocks = 2, threads_per_blocks = 8;
 	int n = num_of_blocks * threads_per_blocks;
	int N = n * n;

	ha = (float*)calloc(N, sizeof(float));
	hb = (float*)calloc(N, sizeof(float));
	CUDA_CHECK_RETURN(hipMalloc((void**)&da, N * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&db, N * sizeof(float)));

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	gTest1<<<dim3(num_of_blocks, num_of_blocks), dim3(threads_per_blocks, threads_per_blocks)>>>(da);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	fprintf(stderr, "%f\n", elapsedTime);


	hipEventRecord(start, 0);
	gTest2<<<dim3(num_of_blocks, num_of_blocks), dim3(threads_per_blocks, threads_per_blocks)>>>(db);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	fprintf(stderr, "%f\n", elapsedTime);

	hipDeviceSynchronize();
	printf("\nGenerated matrix:\n");
	CUDA_CHECK_RETURN(hipMemcpy(ha, da, N * sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < sqrt(N); i++) {
		for (int j = 0; j < sqrt(N); j++) {
			printf("%g  ", ha[j + i * n]);
		}
		printf("\n");
	}
	printf("\n\n");

	transpos<<<dim3(num_of_blocks, num_of_blocks), dim3(threads_per_blocks, threads_per_blocks)>>>(da, db);
	hipDeviceSynchronize();
	printf("\nTransposited matrix:\n");
	hipMemcpy(hb, db, N * sizeof(float), hipMemcpyDeviceToHost);
 	for(int i = 0; i < sqrt(N); i++){
		for(int j = 0; j < sqrt(N); j++){
			printf("%g  ", hb[j + i * n]);
		}
		printf("\n");
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
 	free(ha);
	free(hb);
 	hipFree(da);
	hipFree(db);
 	return 0;
}

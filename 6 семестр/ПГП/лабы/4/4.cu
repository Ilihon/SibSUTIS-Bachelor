#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define CUDA_CHECK_RETURN(value){\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess){\
		fprintf(stderr, "Error %s atline %d in file %s\n",\
		 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}\
}\

// вывод матрицы
void output(float *a, int N){
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++)
			printf("%g  ", a[j + i * N]);
		printf("\n");
	}
	printf("\n\n");
} 

// инициализация матрицы
__global__ void gTest1(float* a){
 	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int I = gridDim.x * blockDim.x;
	a[i + j * I] = (float)(i + j * I);
}

__global__ void gTest2(float *a){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int J = gridDim.y * blockDim.y;
	a[j + i * J] = (float)(threadIdx.y + blockIdx.y * blockDim.y);
}

// простое транспонирование
__global__ void transpos(float *a, float *b){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int I = gridDim.x * blockDim.x;
	b[j + i * I] = a[i + j * I];
}

// наивное использование разделяемой памяти (динамически)
__global__ void transpos21(float *a, float *b){
	extern __shared__ float buffer[];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;

	buffer[threadIdx.y + threadIdx.x * blockDim.y] = a[i + j * N];
	__syncthreads();
	
	i = threadIdx.x + blockIdx.y * blockDim.x;
	j = threadIdx.y + blockIdx.x * blockDim.y;
	
	b[i + j * N] = buffer[threadIdx.x + threadIdx.y * blockDim.x];
}

#define SH_DIM 32

//алгоритм использования разделяемой памяти (статически)
__global__ void transpos22(float *a, float *b){
	__shared__ float buffer[SH_DIM][SH_DIM];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;

	buffer[threadIdx.y][threadIdx.x] = a[i + j * N];
	__syncthreads();

	i = threadIdx.x + blockIdx.y * blockDim.x;
	j = threadIdx.y + blockIdx.x * blockDim.y;
	b[i + j * N] = buffer[threadIdx.x][threadIdx.y];
}

// алгоритм разрешения конфликта банков
__global__ void transpos3(float *a, float *b){
	__shared__ float buffer[SH_DIM][SH_DIM + 1];
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;

	buffer[threadIdx.y][threadIdx.x] = a[i + j * N];
	__syncthreads();
	
	i = threadIdx.x + blockIdx.y * blockDim.x;
	j = threadIdx.y + blockIdx.x * blockDim.y;
	b[i + j * N] = buffer[threadIdx.x][threadIdx.y];
}
 
int main(int argc, char* argv[]){
	if(argc < 3){
		printf("USAGE: matrix <dimension of matrix> <dimension_of_threads\n");
		return -1;
	}
	int N = atoi(argv[1]);
	int dim_of_threads = atoi(argv[2]);
	if(N % dim_of_threads){
		printf("change dimensions\n");
		return -1;
	}
	int dim_of_blocks = N / dim_of_threads;
	const int max_size = 1<<8;
	if(dim_of_blocks > max_size){
		printf("too many blocks\n");
		return -1;
	} 
 	float *da, *h, *db;
 	h = (float*)calloc(N * N, sizeof(float));
	//hb = (float*)calloc(N * N, sizeof(float));
	CUDA_CHECK_RETURN(hipMalloc((void**)&da, N * N * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&db, N * N * sizeof(float)));
	gTest1<<<dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads)>>>(da);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	memset(h, 0.0, N * N * sizeof(float));
	hipMemcpy(h, da, N * N * sizeof(float), hipMemcpyDeviceToHost);
	//output(h, N);
	
	// простое транспонирование
	transpos<<<dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads)>>>(da, db);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	memset(h, 0.0, N * N * sizeof(float));
 	CUDA_CHECK_RETURN(hipMemcpy(h, db, N * N * sizeof(float), hipMemcpyDeviceToHost));
	//output(h, N);

	// наивное разделение памяти (динамически)
	transpos21<<<dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads),dim_of_threads * dim_of_threads * sizeof(float)>>>(da, db);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	memset(h, 0.0, N * N * sizeof(float));
	CUDA_CHECK_RETURN(hipMemcpy(h, db, N * N * sizeof(float), hipMemcpyDeviceToHost));
	//output(h, N);

	// наиваное разделение памяти (статически)
	transpos22<<<dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads)>>>(da,db);
	hipDeviceSynchronize();
	memset(h, 0.0, N * N * sizeof(float));
	hipMemcpy(h, db, N * N * sizeof(float), hipMemcpyDeviceToHost);
	//output(h, N);

	// разрешение конфликтов банков
	transpos3<<<dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads)>>>(da, db);
	hipDeviceSynchronize();
	memset(h, 0.0, N * N * sizeof(float));
	hipMemcpy(h, db, N * N * sizeof(float), hipMemcpyDeviceToHost);
	//output(h, N);
	
 	free(h);
 	hipFree(da);
	hipFree(db);
 	return 0;
}


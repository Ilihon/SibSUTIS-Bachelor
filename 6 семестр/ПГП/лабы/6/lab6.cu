#include "hip/hip_runtime.h"
#include <stdio.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N){
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(){
	hipDeviceProp_t prop;
	int whichDevice;
	
	float elapsedTime;

	int *dev_a, *host_a, *dev_b, *host_b, *dev_c, *host_c;
	//int *host_a_page, *host_b_page, *host_c_page;
	
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	/*
	host_a_page = (int*)calloc(N, sizeof(int));
	host_b_page = (int*)calloc(N, sizeof(int));
	host_c_page = (int*)calloc(N, sizeof(int));
	memset(host_a_page, 0, N * sizeof(float));
	memset(host_b_page, 0, N * sizeof(float));
	memset(host_c_page , 0, N * sizeof(float));
	memset(dev_a, 0, N * sizeof(float));
	memset(dev_b, 0, N * sizeof(float));
	memset(dev_c , 0, N * sizeof(float));
	*/
	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap){
		printf("Device does not support overlapping\n");
		return 0;
	}
	
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N){
		hipMemcpy(host_a + i, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(host_b + i, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);

		kernel <<< N / 2048, 32, 0, stream >>> (dev_a, dev_b, dev_c);
		hipMemcpy(dev_c, host_c + i, N * sizeof(int), hipMemcpyHostToDevice);
	}
	hipStreamSynchronize(stream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time0: %g\n", elapsedTime);
	
	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N){
		hipMemcpyAsync(host_a + i, dev_a, N * sizeof(int), hipMemcpyDeviceToHost, stream);
		hipMemcpyAsync(host_b + i, dev_b, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	
		kernel <<< N / 2048, 32, 0, stream >>> (dev_a, dev_b, dev_c);
		hipMemcpyAsync(dev_c, host_c + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
	}
	
	hipStreamSynchronize(stream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time1: %g\n", elapsedTime);
	
////// I
	
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1; 
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	hipMalloc((void**)&dev_a0, N * sizeof(int));
	hipMalloc((void**)&dev_a1, N * sizeof(int));
	hipMalloc((void**)&dev_b0, N * sizeof(int));
	hipMalloc((void**)&dev_b1, N * sizeof(int));
	hipMalloc((void**)&dev_c0, N * sizeof(int));
	hipMalloc((void**)&dev_c1, N * sizeof(int));
	
	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N * 2){
	///////// Первый поток //////////
		hipMemcpyAsync(host_a + i, dev_a0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_b + i, dev_b0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
	
		kernel <<< N / 2048, 32, 0, stream0 >>> (dev_a0, dev_b0, dev_c0);
		hipMemcpyAsync(dev_c0, host_c + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);

	//////// Второй поток //////////
		hipMemcpyAsync(host_a + i + N, dev_a1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(host_b + i + N, dev_b1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	
		kernel <<< N / 2048, 32, 0, stream1 >>> (dev_a1, dev_b1, dev_c1);
		hipMemcpyAsync(dev_c1, host_c + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
	}
	
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time2: %g\n", elapsedTime);

	
////// II
	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N *2){
	//////// Перемежаемые потоки ///////
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
	
		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

	kernel <<< N / 2048, 32, 0, stream0 >>> (dev_a0, dev_b0, dev_c0);
	kernel <<< N / 2048, 32, 0, stream1 >>> (dev_a1, dev_b1, dev_c1);

	hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
	hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time3: %g\n", elapsedTime);

/////// destroy
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);

	return 0;
}
		 

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <stdio.h>

using namespace thrust;
using namespace std;

#define A 0.2
#define B 0.01


//обычное 
__global__ void kernel(float koef, float *f, float *res)
{
    int cur = threadIdx.x + blockDim.x * blockIdx.x;
    int prev = cur - 1;
    if(prev == -1)
    {
        res[cur] = f[cur];
    }else
    {
        res[cur] = f[cur] + (koef) * (f[prev] - f[cur]);
    }
}

struct functor
{
    const float koef;
    functor(float _koef) : koef(_koef){}
    __host__ __device__ float operator()(float x, float y)
    {
        return x + koef * (y - x);
    }
};



void iteration(float _koef, thrust::device_vector<float>::iterator  
x,thrust::device_vector<float>::iterator xs, thrust::device_vector<float>::iterator 
y)
{
    functor func(_koef);
    thrust::transform(x+1, xs, x, y+1, func);
}

float x_func(float x){
        return x*x * exp(-(x-A)*(x-A)/B);
}

float t_func(float t){
        return 0;
}

int main()
{
    int Nx = 1024;//максимум на осях
    int Nt = 1024;//максимум по времени
    float tlmt = 0.2;

    float dx = 1.0f/Nx;
    float dt = tlmt/Nt;

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    float *x;
    float *t;
    thrust::host_vector<float> thr(Nx*Nt);//создаём вектор матрицу
    float *cda;
	//выделение памяти
    hipHostAlloc((void**)&x, Nx*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**)&t, Nt*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**)&cda, Nt*Nx*sizeof(float), hipHostMallocDefault);
	//заполняем матрицу нулями
    for(int i=0;i<Nx;i++){
        for(int j=0;j<Nt;j++){
                thr[i+j*Nt]=0;
                cda[i+j*Nt]=0;
        }
    }


    float value = 0;
    for(int i=0;i<Nx;i++,value+=dx){
        x[i]=value;
        thr[i+0*Nt]=x_func(x[i]);
        cda[i+0*Nt]=x_func(x[i]);
    }

    value = 0;

    for(int i=0;i<Nt;i++,value+=dt){
        t[i]=value;
        thr[0+i*Nt]=t_func(t[i]);
        cda[0+i*Nt]=t_func(t[i]);
    }

    thrust::device_vector<float> dev(Nx*Nt);

    thrust::copy(thr.begin(), thr.end(),dev.begin());

    functor func(dt/dx);

    hipEventRecord(start,0);
    for(int j=0;j<Nt-1;j++){
        thrust::transform(dev.begin()+(j*Nx)+1, dev.begin()+((j+1)*Nx), dev.begin()+(j*Nx), dev.begin() +((j+1)*Nx)+1, func);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    thrust::copy(dev.begin(),dev.end(),thr.begin());
    hipEventElapsedTime(&time, start, stop);
    printf("Thrust time: %f ms\n", time);
	/*for (int i = 0; i < (1 << 8); i++) {
		printf("%d\t%g\n", i, thr[i]);
	}*/
//...................................................................
    
    float *dev_cda;


    hipMalloc((void **)&dev_cda, Nx * Nt * sizeof(float));
    hipMemcpy(dev_cda, cda, Nx*Nt * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    for(int i=0;i<Nt-1;i++){
        kernel <<< 1, Nx >>> (dt/dx, dev_cda + (i*Nx), dev_cda + ((i+1)*Nx) );
        hipDeviceSynchronize();
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipMemcpy(cda, dev_cda, Nx*Nt * sizeof(float), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&time, start, stop);
    printf("\nCUDA time: %f ms\n\n", time);

    hipFree(dev_cda);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}

#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <thrust/fill.h>
#include <thrust/sequence.h>

using namespace thrust;

struct saxpy_functor
{
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__ float operator()(float x, float y){
		return a * x + y;
	}
};

void saxpy(float a, thrust::device_vector<float>& x, thrust::device_vector<float>& y){
	saxpy_functor func(a);
	
	transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

__global__ void cuda_saxpy(float *a, float *b, float alpha){
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	
	a[j] = j;
	b[j] = 0.87; 
	a[j] = alpha * a[j] + b[j];
}

int main(){
	float elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	thrust::host_vector<float> h1(1 << 24);
	thrust::host_vector<float> h2(1 << 24);
	thrust::sequence(h1.begin(), h1.end());
	thrust::fill(h2.begin(), h2.end(), 0.87);

	thrust::device_vector<float> d1 = h1;
	thrust::device_vector<float> d2 = h2;
	
	hipEventRecord(start, 0);	
	saxpy(3.0, d1, d2);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Thrust time: %g\n", elapsedTime);

	h2 = d2;
	h1 = d1;

	for(int i = 0; i < (1 << 8); i++){
		printf("%d\t%g\t%g\n", i, h1[i], h2[i]);
	}
	
	float *da, *db, *h, alpha = 3.0;
	int threads_per_block = 512, N = 1 << 24;
	int num_of_blocks = N / threads_per_block;

	h = (float*)calloc(N, sizeof(float));

	hipMalloc((void**)&da, N * sizeof(float));
	hipMalloc((void**)&db, N * sizeof(float));
	
	hipEventRecord(start, 0);
	cuda_saxpy<<<dim3(num_of_blocks), dim3(threads_per_block)>>>(da, db, alpha);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);	
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Cuda time: %g\n", elapsedTime);

	hipMemcpy(h, da, N * sizeof(float), hipMemcpyDeviceToHost);

	//for(int i = 0; i < (1 << 8); i++){
	//	printf("%g\n", h[i]);
	//}
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(h);
	hipFree(da);
	hipFree(db);
	return 0;
}


#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <thrust/fill.h>
#include <thrust/sequence.h>

#pragma comment (lib, "cublas.lib")
#include <hipblas.h>

struct saxpy_functor{
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__ float operator()(float x, float y){
		return a * x + y;
	}
};

void saxpy(float a, thrust::device_vector<float>& x, thrust::device_vector<float>& y){
	saxpy_functor func(a);
	thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

__host__ void print_array(float *data1, float *data2, int num_elem, const char *prefix){
	printf("\n%s", prefix);
	for(int i = 0; i < num_elem; i++)
		printf("\n%2d: %2.4f %2.4f", i + 1, data1[i], data2[i]);
}

__global__ void cuda_saxpy(float *a, float *b, float alpha){
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	a[j] = j;
	b[j] = 0.87; 
	a[j] = alpha * a[j] + b[j];
}

int main(){
	float elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	thrust::host_vector<float> h1(1 << 24);
	thrust::host_vector<float> h2(1 << 24);
	thrust::sequence(h1.begin(), h1.end());
	thrust::fill(h2.begin(), h2.end(), 0.87);

	thrust::device_vector<float> d1 = h1;
	thrust::device_vector<float> d2 = h2;
	
	hipEventRecord(start, 0);	
	saxpy(3.0, d1, d2);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Saxpy time: %g\n", elapsedTime);
		
	const int num_elem = 8;
	const size_t size_in_bytes = (num_elem * sizeof(float));
	float *da, *db, *ha, *hb;

	hipMalloc((void**) &da, size_in_bytes);
	hipMalloc((void**) &db, size_in_bytes);

	hipHostMalloc((void**) &ha, size_in_bytes);
	hipHostMalloc((void**) &hb, size_in_bytes);

	memset(ha, 0, size_in_bytes);
	memset(hb, 0, size_in_bytes);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	for (int i = 0; i < num_elem; i++) {
		ha[i] = (float)i;
		hb[i] = 0.87;
	}

	print_array(ha, hb, num_elem, "Before set");
	printf("\n");

	const int num_rows = num_elem;
	const int num_cols = 1;
	const size_t elem_size = sizeof(float);

	hipblasSetMatrix(num_rows, num_cols, elem_size, ha, num_rows, da, num_rows);
	hipblasSetMatrix(num_rows, num_cols, elem_size, hb, num_rows, db, num_rows);

	const int stride = 1;
	float alpha = 3.0F;
	
	hipEventRecord(start, 0);
	hipblasSaxpy(cublas_handle, num_elem, &alpha, da, stride, db, stride);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipblasGetMatrix(num_rows, num_cols, elem_size, da, num_rows, ha, num_rows);
	hipblasGetMatrix(num_rows, num_cols, elem_size, db, num_rows, hb, num_rows);

	const int default_stream = 0;
	hipStreamSynchronize(default_stream);

	print_array(ha, hb, num_elem, "After set");
	printf("\n");
	
/*
	hipEventRecord(start, 0);
	cuda_saxpy<<<dim3(num_of_blocks), dim3(threads_per_block)>>>(da, db, alpha);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);	

	printf("Cuda Saxpy time: %g\n", elapsedTime);
*/
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Cublas time: %g\n", elapsedTime);

	//for(int i = 0; i < (1 << 8); i++){
	//	printf("%g\t%g\n", ha[i], hb[i]);
	//}
	
	hipblasDestroy(cublas_handle);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipHostFree(ha);
	hipHostFree(hb);
	hipFree(da);
	hipFree(db);
	return 0;
}


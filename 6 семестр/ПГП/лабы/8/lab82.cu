
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>

#pragma comment (lib, "cufft.lib")
#include <hipfft/hipfft.h>

#define NX 365
#define BATCH 1
#define pi 3.141592

__global__ void gInitData(hipfftComplex *data){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	float x = i * 2.0F * pi / NX;
	data[i].x = cosf(x) - 3.0F * sinf(x);
	data[i].y = 0.0F;
}

int main(){
	hipfftHandle plan;
	hipfftComplex *data;
	hipfftComplex *data_h = (hipfftComplex*)calloc(NX, sizeof(hipfftComplex));
	
	hipMalloc((void**) &data, sizeof(hipfftComplex) * NX * BATCH);

	FILE *f = fopen("file.txt", "r");
	for (int i = 0; i < NX; i++){
		float a, b, c;
		fscanf(f, "%f%f%f%f", &a, &b, &data_h[i].x, &c);
		if (data_h[i].x == 999 && i != 0)
			data_h[i].x = data_h[i-1].x;
		data_h[i].y = 0;
	}		

	hipMemcpy(data, data_h, NX * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	
	if (hipGetLastError() != hipSuccess){
		printf("Cuda error: Failed to allocate\n");
		return -1;
	}

	//gInitData <<< 1, NX >>> (data);
	//cudaDeviceSynchronize();

	if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		printf("CUFFT error: Plan creation failed\n");
		return -1;
	}

	if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		printf("CUFFT error: EXECC2C Forward failed\n");
		return -1;
	}

	if (hipDeviceSynchronize() != hipSuccess){
		printf("Cuda error: failed to synchronize\n");
		return -1;
	}

	hipMemcpy(data_h, data, NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < NX; i++)
		printf("%g    \t%g\n", data_h[i].x, data_h[i].y);
	
	hipfftDestroy(plan);
	hipFree(data);
	free(data_h);

	return 0;
} 

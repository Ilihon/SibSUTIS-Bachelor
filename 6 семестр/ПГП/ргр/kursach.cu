#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <hipblas.h>

#pragma comment (lib, "cublas.lib")
#include <hipblas.h>
#pragma comment (lib, "cufft.lib")
#include <hipfft/hipfft.h>

using namespace thrust;
using namespace std;

struct saxpy_functor
{
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__ float operator()(float x, float y) {
		return a * x + y;
	}
};

void saxpy(float a, thrust::device_vector<float>& x, thrust::device_vector<float>& y){
	saxpy_functor func(a);
	thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

__global__ void cuda_saxpy(float *a, float *b, float alpha){
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	a[j] = j;
	b[j] = 0.87; 
	a[j] = alpha * a[j] + b[j];
}

__host__ void print_array(float *data1, float *data2, int num_elem, const char *prefix) {
	printf("\n%s", prefix);
	for (int i = 0; i < num_elem; i++)
		printf("\n%2d: %2.4f %2.4f", i + 1, data1[i], data2[i]);
}
 
int main() {
	float elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Trust 
	thrust::host_vector<float> h1(1 << 25);
	thrust::host_vector<float> h2(1 << 25);
	thrust::sequence(h1.begin(), h1.end());
	thrust::fill(h2.begin(), h2.end(), 0.87);

	thrust::device_vector<float> d1 = h1;
	thrust::device_vector<float> d2 = h2;

	hipEventRecord(start, 0);
	saxpy(3.0, d1, d2);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Trust time: %g\n", elapsedTime);
	printf("\n");

	h2 = d2;
	h1 = d1;
	/*
	for (int i = 0; i < 8; i++) {
		printf("%d\t%g\t%g\n", i, h1[i], h2[i]);
	}
	*/

	// Сырой Cuda
	float *h, *da, *db, alpha = 3.0F;
	int threads_per_block = 525, N = 1 << 25;
	int num_of_blocks = N / threads_per_block;

	h = (float*)calloc(N, sizeof(float));

	hipMalloc((void**)&da, N * sizeof(float));
	hipMalloc((void**)&db, N * sizeof(float));

	hipEventRecord(start, 0);
	cuda_saxpy << <dim3(num_of_blocks), dim3(threads_per_block) >> > (da, db, alpha);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("\nCuda time: %g\n", elapsedTime);

	hipMemcpy(h, da, N * sizeof(float), hipMemcpyDeviceToHost);
	/*
	for (int i = 0; i < 8; i++) {
		printf("%g\n", h[i]);
	}
	*/
	// Cublas
	const int num_elem = 1 << 25;
	const size_t size_in_bytes = (num_elem * sizeof(float));
	float *ha, *hb;

	hipMalloc((void**)&da, size_in_bytes);
	hipMalloc((void**)&db, size_in_bytes);

	hipHostMalloc((void**)&ha, size_in_bytes);
	hipHostMalloc((void**)&hb, size_in_bytes);

	memset(ha, 0, size_in_bytes);
	memset(hb, 0, size_in_bytes);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	for (int i = 0; i < num_elem; i++) {
		ha[i] = (float)i;
		hb[i] = 0.87;
	}

	//print_array(ha, hb, num_elem, "Before set");

	const int num_rows = num_elem;
	const int num_cols = 1;
	const size_t elem_size = sizeof(float);

	hipblasSetMatrix(num_rows, num_cols, elem_size, ha, num_rows, da, num_rows);
	hipblasSetMatrix(num_rows, num_cols, elem_size, hb, num_rows, db, num_rows);

	const int stride = 1;
	alpha = 3.0F;
	
	hipEventRecord(start, 0);
	hipblasSaxpy(cublas_handle, num_elem, &alpha, da, stride, db, stride);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipblasGetMatrix(num_rows, num_cols, elem_size, da, num_rows, ha, num_rows);
	hipblasGetMatrix(num_rows, num_cols, elem_size, db, num_rows, hb, num_rows);

	const int default_stream = 0;
	hipStreamSynchronize(default_stream);
	
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Cublas time: %g\n", elapsedTime);
	printf("\n");
	
	//print_array(ha, hb, num_elem, "After set");
	
	hipblasDestroy(cublas_handle);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipHostFree(ha);
	hipHostFree(hb);
	hipFree(da);
	hipFree(db);
	free(h);
	return 0;
}

